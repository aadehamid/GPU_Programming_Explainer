#include "hip/hip_runtime.h"
// This program demonstrates vector addition using CUDA on the GPU.
// It adds two arrays element-wise in parallel on the GPU and retrieves the result on the CPU.
// Each step is commented for educational clarity.
#include <hip/hip_runtime.h>
#include <>

// C++ specific headers
#include <iostream>

// CUDA kernel function to add two arrays element-wise
// Each thread computes one element of the result array
__global__ void vectorized(int a[], int b[], int c[], size_t size){
    // Calculate the global index for this thread
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    // Make sure we don't go out of bounds
    if (i < size){
        // Perform the addition for element i
        c[i] = a[i] + b[i];
    }
}

int main(){
    // Number of elements in our arrays
    size_t size = 4;

    // Host arrays (CPU memory)
    int a[size]{1,2,3,4}; // First input array
    int b[size]{5,6,7,8}; // Second input array
    int c[size]{0};       // Output array to hold results

    // Device pointers (GPU memory)
    int* cuda_a = nullptr;
    int* cuda_b = nullptr;
    int* cuda_c = nullptr;

    // Allocate memory on the GPU for each array
    hipMalloc(&cuda_a, sizeof(int) * size);
    hipMalloc(&cuda_b, sizeof(int) * size);
    hipMalloc(&cuda_c, sizeof(int) * size);

    // Copy input data from CPU (host) to GPU (device)
    hipMemcpy(cuda_a, a, sizeof(int) * size, hipMemcpyHostToDevice);
    hipMemcpy(cuda_b, b, sizeof(int) * size, hipMemcpyHostToDevice);

    // Launch the CUDA kernel with one block of 'size' threads
    // Each thread will compute one element of the output array
    vectorized<<<1, size>>>(cuda_a, cuda_b, cuda_c, size);
    hipDeviceSynchronize(); // Wait for GPU to finish

    // Copy the result from GPU back to CPU
    hipMemcpy(c, cuda_c, sizeof(int) * size, hipMemcpyDeviceToHost);

    // Print the result array
    std::cout << "Result of vector addition: ";
    for (size_t i{0}; i < size; i++){
        std::cout << c[i] << "  ";
    }
    std::cout << std::endl;

    // Free the GPU memory
    hipFree(cuda_a);
    hipFree(cuda_b);
    hipFree(cuda_c);

    return 0;

}
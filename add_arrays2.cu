#include "hip/hip_runtime.h"
//Declare required CUDA headers
#include <hip/hip_runtime.h>
#include <>

// C++ specific headers
#include <iostream>

// Function declarations
__global__ void vectorized(int a[], int b[], int c[], size_t size){
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < size){
    c[i] = a[i] + b[i];
    }
return;
}

int main(){
    size_t size = 4;
    int a[size]{1,2,3,4};
    int b[size]{5,6,7,8};
    int c[size]{0};

    // Set up pointers to hold memory on GPU
    int* cuda_a = nullptr;
    int* cuda_b = nullptr;
    int* cuda_c = nullptr;

    // Allocate the memory on GPU
    hipMalloc(&cuda_a, sizeof(int) * size);
    cudaMaloc(&cuda_b, sizeof(int) * size);
    hipMalloc(&cuda_c, sizeof(int) * size);

    // Move data from HOST to DEVICE
    hipMemcpy(cuda_a, a, sizeof(a), hipMemcpyHostToDevice)
    hipMemcpy(cuda_b, sizeof(b), hipMemcpyHostToDevice);


    // Call the function on GPU
    vectorized<<<1, sizeof(a)/sizeof(int)>>>(cuda_a, cuda_b, cuda_c, size);
    hipDeviceSynchronize();

    // Move data from GPU to CPU
    hipMemcpy(c, cuda_c, sizeof(c), hipMemcpyDeviceToHost);

    // Print the result
    for (size_t i{0}; i < size; i++){

    std::cout<< c[i] << "  ";
    }

    // Free GPU Memory
    hipFree(cuda_a);
    hipFree(cuda_b);
    hipFree(cuda_c);


}